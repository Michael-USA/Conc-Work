#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>

__global__ void helloWorldKernel() {
    int blockId = blockIdx.x + blockIdx.y * gridDim.x;
    int threadId = blockId * (blockDim.x * blockDim.y) + (threadIdx.y * blockDim.x) + threadIdx.x;

}

int main() {
    // Configure the number of blocks and threads per block
    dim3 blocks(2, 1);   // Use 2 blocks
    dim3 threads(2, 1);  // Each block has 2 threads

    // Launch the kernel
    helloWorldKernel<<<blocks, threads>>>();

    // Wait for GPU to finish before accessing on host
    hipDeviceSynchronize();

    // Check for any errors launching the kernel
    hipError_t cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, Kernel launch failed: %sn, hipGetErrorString(cudaStatus));
        return 1;
    }

    return 0;
}
